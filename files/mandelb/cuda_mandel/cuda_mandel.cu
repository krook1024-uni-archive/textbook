#include "hip/hip_runtime.h"
// Copyright (C) 2019
// Norbert Batfai, batfai.norbert@inf.unideb.hu
// Released under GNU GPLv3

#include <png++/image.hpp>
#include <png++/rgb_pixel.hpp>
#include <sys/times.h>
#include <iostream>

#define SIZE 600
#define ITERATION_LIMIT 32000

// Vegigzongorazza a CUDA a szelesseg x magassag racsot:
__device__ int mandel(int k, int j)
{
	// most eppen a j. sor k. oszlopaban vagyunk

	float a = -2.0, b = .7, c = -1.35, d = 1.35;
	int width = SIZE, height = SIZE, iterationLimit = ITERATION_LIMIT;

	float dx = (b - a) / width;
	float dy = (d - c) / height;
	float reC, imC, reZ, imZ, ujreZ, ujimZ;
	int iteration = 0;

	reC = a + k * dx;
	imC = d - j * dy;
	reZ = 0.0;
	imZ = 0.0;
	iteration = 0;

	while (reZ * reZ + imZ * imZ < 4 && iteration < iterationLimit) {
		ujreZ = reZ * reZ - imZ * imZ + reC;
		ujimZ = 2 * reZ * imZ + imC;
		reZ = ujreZ;
		imZ = ujimZ;

		++iteration;

	}
	return iteration;
}

__global__ void mandelkernel(int *buffer)
{

	int tj = threadIdx.x;
	int tk = threadIdx.y;

	int j = blockIdx.x * 10 + tj;
	int k = blockIdx.y * 10 + tk;

	buffer[j + k * SIZE] = mandel(j, k);

}

void cudamandel(int buffer[SIZE][SIZE])
{

	int *deviceImageBuffer;
	hipMalloc((void **)&deviceImageBuffer, SIZE * SIZE * sizeof(int));

	dim3 grid(SIZE / 10, SIZE / 10);
	dim3 tgrid(10, 10);
	mandelkernel <<< grid, tgrid >>> (deviceImageBuffer);

	hipMemcpy(buffer, deviceImageBuffer, SIZE * SIZE * sizeof(int),
		   hipMemcpyDeviceToHost);
	hipFree(deviceImageBuffer);

}

int main(int argc, char *argv[])
{

	// Merunk idot (PP 64)
	clock_t delta = clock();
	// Merunk idot (PP 66)
	struct tms tmsbuf1, tmsbuf2;
	times(&tmsbuf1);

	int buffer[SIZE][SIZE];

	cudamandel(buffer);

	png::image < png::rgb_pixel > image(SIZE, SIZE);

	for (int j = 0; j < SIZE; ++j) {
		//sor = j;
		for (int k = 0; k < SIZE; ++k) {
			image.set_pixel(k, j,
					png::rgb_pixel(255 -
						       (255 * buffer[j][k]) /
						       ITERATION_LIMIT,
						       255 -
						       (255 * buffer[j][k]) /
						       ITERATION_LIMIT,
						       255 -
						       (255 * buffer[j][k]) /
						       ITERATION_LIMIT));
		}
	}

	image.write("mandel.png");

	times(&tmsbuf2);
	std::cout << tmsbuf2.tms_utime - tmsbuf1.tms_utime + tmsbuf2.tms_stime -
	    tmsbuf1.tms_stime << std::endl;

	delta = clock() - delta;
	std::cout << (float)delta / CLOCKS_PER_SEC << " sec" << std::endl;

}
